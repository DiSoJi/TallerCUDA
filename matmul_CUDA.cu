#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include <hip/hip_runtime.h>

#define SIZE 4

__global__ 
void matrixMultiplicationKernel(int* A, int* B, int* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    int tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}
/*
void cudaCheckError() {
    hipError_t e=hipGetLastError();
    if(e!=hipSuccess) {
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
      exit(0); 
    }
   }
   */
int main (){
    
    //I'm using vectors with 16 elements to represent the matrix (4 rows with 4 values)
    int *first = (int*) malloc(SIZE * SIZE * sizeof(int)); 
    int *second = (int*) malloc(SIZE * SIZE * sizeof(int));
    int *result = (int*) malloc(SIZE * SIZE * sizeof(int));

    //Fill local data:
    for (int i = 0; i < SIZE; i++){
        for (int j = 0; j < SIZE; j++){
            //i+j gives the position (columb) in the row, for each row.
            first[i+j] = i; 
            second[i+j] = j;
        }
    }

    int *first_gpu;
    int *second_gpu;
    int *result_gpu;

    //Iterations
    int N=SIZE; //size of vector
    //Number of blocks
    int nBytes = N*N*sizeof(int);
    //Block size and number
    int block_size, block_no;
    block_size = 250; //threads per block
    block_no = N/block_size;

    printf("Allocating device memory on host..\n");
    //GPU memory allocation
    hipMalloc((void **) &first_gpu,  nBytes);
    hipMalloc((void **) &second_gpu, nBytes);
    hipMalloc((void **) &result_gpu, nBytes);

    //Work definition////////////////////
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);
    /////////////////////////////////////
    printf("Copying to device..\n");
    hipMemcpy(first_gpu, first, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(second_gpu, second, nBytes, hipMemcpyHostToDevice);

    clock_t start_d=clock();
    printf("Doing GPU Matrix Multiplication\n");
    matrixMultiplicationKernel<<<block_no,block_size>>>(first_gpu, second_gpu, result_gpu, N);
    //cudaCheckError();
    clock_t end_d = clock();
    //Wait for kernel call to finish
    hipDeviceSynchronize();
    
    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(result, result_gpu, nBytes, hipMemcpyDeviceToHost);
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    printf("Time it took on GPU: %f", time_d);
    //Free GPU memory
    hipFree(first_gpu);
    hipFree(second_gpu);
    hipFree(result_gpu);
    return 0;
}